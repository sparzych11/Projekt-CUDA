#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <iostream>
#include <>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

const int numBins = 256; 

//Funkcja do obliczania Histogramu:
__global__
void computeHistogram(const unsigned char* image, int width, int height, int* histogram) {

    __shared__ int sharedHistogram[numBins];

    for (int i = threadIdx.x; i < numBins; i += blockDim.x) {
        sharedHistogram[i] = 0;
    }
    __syncthreads();

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (idx < width * height) {
        atomicAdd(&sharedHistogram[image[idx]], 1);
        idx += stride;
    }
    __syncthreads();

    for (int i = threadIdx.x; i < numBins; i += blockDim.x) {
        atomicAdd(&histogram[i], sharedHistogram[i]);
    }

}

//Funkcja do liniowej modyfikacji Histogramu:
__global__
void linearTransform(const unsigned char* inputImage, unsigned char* outputImage, int width, int height, int* histogram) {
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (idx < width * height * 3) {
        float scalingFactor = 255.0 / (width * height);
        outputImage[idx] = static_cast<unsigned char>(histogram[inputImage[idx]] * scalingFactor);
        idx += stride;
    }
}
//Funkcja do wyrównania Histogramu:
__global__
void histogramEqualization(const unsigned char* inputImage, unsigned char* outputImage, int width, int height, int* histogram) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (idx < width * height * 3) {
        float cdf = 0.0;
        for (int i = 0; i <= inputImage[idx]; ++i) {
            cdf += histogram[i];
        }

        outputImage[idx] = static_cast<unsigned char>((cdf / (width * height * 3)) * 255.0);

        idx += stride;
    }
}

//Funkcja do rozszerzenia liniowego Histogramu:
__global__
void linearStretch(const unsigned char* inputImage, unsigned char* outputImage, int width, int height, int* histogram) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    int minValue = 0;
    int maxValue = numBins - 1;
    while (histogram[minValue] == 0) ++minValue;
    while (histogram[maxValue] == 0) --maxValue;

    while (idx < width * height * 3) {
        outputImage[idx] = static_cast<unsigned char>((inputImage[idx] - minValue) * (255.0 / (maxValue - minValue)));
        idx += stride;
    }
}
//Funkcja do rozszerzenia nieliniowego Histogramu:
__global__
void nonlinearStretch(const unsigned char* inputImage, unsigned char* outputImage, int width, int height, int* histogram) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    int minValue = 0;
    int maxValue = numBins - 1;
    while (histogram[minValue] == 0) ++minValue;
    while (histogram[maxValue] == 0) --maxValue;

    while (idx < width * height * 3) {
        outputImage[idx] = static_cast<unsigned char>(
            255.0 * sqrt((inputImage[idx] - minValue) / static_cast<float>(maxValue - minValue))
            );
        idx += stride;
    }
}

// Funkcja do progowania Histogramu:
__global__
void thresholdHistogram(const unsigned char* inputImage, unsigned char* outputImage, int width, int height, int* histogram, unsigned char threshold) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (idx < width * height * 3) {
        outputImage[idx] = (inputImage[idx] > threshold) ? 255 : 0;
        idx += stride;
    }
}

// Funkcja do inwersji Histogramu:
__global__
void invertHistogram(const unsigned char* inputImage, unsigned char* outputImage, int width, int height, int* histogram) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (idx < width * height * 3) {
        outputImage[idx] = 255 - inputImage[idx];
        idx += stride;
    }
}

int main() {
    // Wczytanie obrazu:
    int width, height, channels;
    unsigned char* image = stbi_load("test.jpg", &width, &height, &channels, 0);

    if (!image) {
        std::cerr << "Failed to load image." << std::endl;
        return 1;
    }

    // Wyznaczenie Histogramu:
    int histogram[numBins] = { 0 };
    unsigned char* d_image;
    int* d_histogram;
    hipMalloc((void**)&d_image, width * height * channels);
    hipMalloc((void**)&d_histogram, numBins * sizeof(int));
    hipMemcpy(d_image, image, width * height * channels, hipMemcpyHostToDevice);
    hipMemset(d_histogram, 0, numBins * sizeof(int));
    int blockSize = 256;
    int numBlocks = (width * height + blockSize - 1) / blockSize;
    computeHistogram << <numBlocks, blockSize >> > (d_image, width, height, d_histogram);
    hipMemcpy(histogram, d_histogram, numBins * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < numBins; ++i) {
        std::cout << "Bin " << i << ": " << histogram[i] << std::endl;
    }

    // Operacja liniowa na Histogramie:
    unsigned char* d_outputImage;
    hipMalloc((void**)&d_outputImage, width * height * channels);
    linearTransform << <numBlocks, blockSize >> > (d_image, d_outputImage, width, height, d_histogram);
    unsigned char* outputImage = new unsigned char[width * height * channels];
    hipMemcpy(outputImage, d_outputImage, width * height * channels, hipMemcpyDeviceToHost);
    stbi_write_jpg("test_linear.jpg", width, height, channels, outputImage, 100);

    // Wyrównanie Histogramu:
    unsigned char* d_equalizedImage;
    hipMalloc((void**)&d_equalizedImage, width * height * channels);
    histogramEqualization << <numBlocks, blockSize >> > (d_image, d_equalizedImage, width, height, d_histogram);
    unsigned char* equalizedImage = new unsigned char[width * height * channels];
    hipMemcpy(equalizedImage, d_equalizedImage, width * height * channels, hipMemcpyDeviceToHost);
    stbi_write_jpg("equal_test.jpg", width, height, channels, equalizedImage, 100);

    // Rozszerzenie liniowe Histogramu:
    unsigned char* d_stretchedImage;
    hipMalloc((void**)&d_stretchedImage, width * height * channels);
    linearStretch << <numBlocks, blockSize >> > (d_image, d_stretchedImage, width, height, d_histogram);
    unsigned char* stretchedImage = new unsigned char[width * height * channels];
    hipMemcpy(stretchedImage, d_stretchedImage, width * height * channels, hipMemcpyDeviceToHost);
    stbi_write_jpg("stretched_test.jpg", width, height, channels, stretchedImage, 100);

    // Rozszerzenie nieliniowe Histogramu:
    unsigned char* d_nonlinearStretchedImage;
    hipMalloc((void**)&d_nonlinearStretchedImage, width * height * channels);
    nonlinearStretch << <numBlocks, blockSize >> > (d_image, d_nonlinearStretchedImage, width, height, d_histogram);
    unsigned char* nonlinearStretchedImage = new unsigned char[width * height * channels];
    hipMemcpy(nonlinearStretchedImage, d_nonlinearStretchedImage, width * height * channels, hipMemcpyDeviceToHost);
    stbi_write_jpg("stretched_nonlinear_test.jpg", width, height, channels, nonlinearStretchedImage, 100);

    // Progowanie Histogramu:
    unsigned char* d_thresholdImage;
    hipMalloc((void**)&d_thresholdImage, width * height * channels);
    unsigned char* thresholdImage = new unsigned char[width * height * channels];
    unsigned char thresholdValue = 128; // Przykładowy próg
    thresholdHistogram << <numBlocks, blockSize >> > (d_image, d_thresholdImage, width, height, d_histogram, thresholdValue);
    hipMemcpy(thresholdImage, d_thresholdImage, width * height * channels, hipMemcpyDeviceToHost);
    stbi_write_jpg("threshold_test.jpg", width, height, channels, thresholdImage, 100);

    // Inwersja Histogramu:
    unsigned char* d_invertedImage;
    hipMalloc((void**)&d_invertedImage, width * height * channels);
    unsigned char* invertedImage = new unsigned char[width * height * channels];
    invertHistogram << <numBlocks, blockSize >> > (d_image, d_invertedImage, width, height, d_histogram);
    hipMemcpy(invertedImage, d_invertedImage, width * height * channels, hipMemcpyDeviceToHost);
    stbi_write_jpg("inverted_test.jpg", width, height, channels, invertedImage, 100);

    // Zwolnienie pamięci GPU:
    hipFree(d_image);
    hipFree(d_histogram);
    hipFree(d_outputImage);
    hipFree(d_equalizedImage);
    hipFree(d_stretchedImage);
    hipFree(d_nonlinearStretchedImage);
    hipFree(d_thresholdImage);
    hipFree(d_invertedImage);

    // Zwolnienie pamięci CPU:
    stbi_image_free(image);
    delete[] outputImage;
    delete[] equalizedImage;
    delete[] stretchedImage;
    delete[] nonlinearStretchedImage;
    delete[] thresholdImage;
    delete[] invertedImage;
    return 0;
}